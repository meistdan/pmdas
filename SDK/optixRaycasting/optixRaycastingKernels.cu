#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_runtime.h>

#include "optixRaycastingKernels.h"

#include <sutil/vec_math.h>


inline int idivCeil( int x, int y )
{
    return ( x + y - 1 ) / y;
}


__global__ void createRaysOrthoKernel( Ray* rays, int width, int height, float x0, float y0, float z, float dx, float dy )
{
    const int rayx = threadIdx.x + blockIdx.x * blockDim.x;
    const int rayy = threadIdx.y + blockIdx.y * blockDim.y;
    if( rayx >= width || rayy >= height )
        return;

    const int idx    = rayx + rayy * width;
    rays[idx].origin = make_float3( x0 + rayx * dx, y0 + rayy * dy, z );
    rays[idx].tmin   = 0.0f;
    rays[idx].dir    = make_float3( 0, 0, 1 );
    rays[idx].tmax   = 1e34f;
}


// Note: uses left handed coordinate system
void createRaysOrthoOnDevice( Ray* rays_device, int width, int height, float3 bbmin, float3 bbmax, float padding )
{
    const float3 bbspan = bbmax - bbmin;
    float        dx     = bbspan.x * ( 1 + 2 * padding ) / width;
    float        dy     = bbspan.y * ( 1 + 2 * padding ) / height;
    float        x0     = bbmin.x - bbspan.x * padding + dx / 2;
    float        y0     = bbmin.y - bbspan.y * padding + dy / 2;
    float        z      = bbmin.z - fmaxf( bbspan.z, 1.0f ) * .001f;

    dim3 blockSize( 32, 16 );
    dim3 gridSize( idivCeil( width, blockSize.x ), idivCeil( height, blockSize.y ) );
    createRaysOrthoKernel<<<gridSize, blockSize>>>( rays_device, width, height, x0, y0, z, dx, dy );
}


__global__ void translateRaysKernel( Ray* rays, int count, float3 offset )
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if( idx >= count )
        return;

    rays[idx].origin = rays[idx].origin + offset;
}


void translateRaysOnDevice( Ray* rays_device, int count, float3 offset )
{
    const int blockSize  = 512;
    const int blockCount = idivCeil( count, blockSize );
    translateRaysKernel<<<blockCount, blockSize>>>( rays_device, count, offset );
}


__global__ void shadeHitsKernel( float3* image, int count, const Hit* hits )
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if( idx >= count )
        return;

    const float3 backgroundColor = make_float3( 0.2f, 0.2f, 0.2f );
    if( hits[idx].t < 0.0f )
    {
        image[idx] = backgroundColor;
    }
    else
    {
        image[idx] = 0.5f * hits[idx].geom_normal + make_float3( 0.5f, 0.5f, 0.5f );
    }
}


void shadeHitsOnDevice( float3* image_device, int count, const Hit* hits_device )
{
    const int blockSize  = 512;
    const int blockCount = idivCeil( count, blockSize );
    shadeHitsKernel<<<blockCount, blockSize>>>( image_device, count, hits_device );
}


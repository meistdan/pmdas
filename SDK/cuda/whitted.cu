#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include <cuda/LocalGeometry.h>
#include <cuda/helpers.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>

#include "whitted_cuda.h"

#define M_PI       3.14159265358979323846   // pi
#define M_PI_2     1.57079632679489661923   // pi/2
#define M_PI_4     0.785398163397448309616  // pi/4

//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__pinhole()
{
    const uint3  launch_idx     = optixGetLaunchIndex();
    const uint3  launch_dims    = optixGetLaunchDimensions();
    const float3 eye            = whitted::params.eye;
    const float3 U              = whitted::params.U;
    const float3 V              = whitted::params.V;
    const float3 W              = whitted::params.W;
    const int    subframe_index = whitted::params.subframe_index;

    //
    // Generate camera ray
    //
    unsigned int seed = tea<4>( launch_idx.y * launch_dims.x + launch_idx.x, subframe_index );

    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter =
        subframe_index == 0 ? make_float2( 0.5f, 0.5f ) : make_float2( rnd( seed ), rnd( seed ) );

    const float2 d =
        2.0f
            * make_float2( ( static_cast<float>( launch_idx.x ) + subpixel_jitter.x ) / static_cast<float>( launch_dims.x ),
                           ( static_cast<float>( launch_idx.y ) + subpixel_jitter.y ) / static_cast<float>( launch_dims.y ) )
        - 1.0f;
    float3 ray_direction = normalize(make_float3(d.x, d.y, 1.0f));
    float3 ray_origin = make_float3(0.0f);

    //
    // Depth of field
    // 
    if (whitted::params.lens_radius > 0.0f)
    {
        // Map uniform random numbers to $[-1,1]^2$
        float2 rnd2 = make_float2(rnd(seed), rnd(seed));
        float2 offset = 2.f * rnd2 - make_float2(1, 1);

        // Handle degeneracy at the origin
        float2 lens = make_float2(0.0f);
        if (offset.x != 0 || offset.y != 0)
        {
            // Apply concentric mapping to point
            float theta, r;
            if (fabs(offset.x) > fabs(offset.y)) 
            {
                r = offset.x;
                theta = M_PI_4 * (offset.y / offset.x);
            }
            else 
            {
                r = offset.y;
                theta = M_PI_2 - M_PI_4 * (offset.x / offset.y);
            }
            lens = whitted::params.lens_radius * r * make_float2(cosf(theta), sinf(theta));
        }

        // Compute point on plane of focus
        float ft = whitted::params.focal_distance / ray_direction.z;
        float3 focus = ft * ray_direction;

        // Update ray for effect of lens
        ray_origin = make_float3(lens.x, lens.y, 0);
        ray_direction = normalize(focus - ray_origin);
    }

    // Transform
    ray_origin += eye;
    ray_direction = normalize(ray_direction.x * U + ray_direction.y * V + ray_direction.z * W);

    //
    // Trace camera ray
    //
    whitted::PayloadRadiance payload;
    payload.result     = make_float3( 0.0f );
    payload.importance = 1.0f;
    payload.depth      = 0.0f;

    traceRadiance( whitted::params.handle, ray_origin, ray_direction,
                   0.01f,  // tmin       // TODO: smarter offset
                   1e16f,  // tmax
                   &payload );

    //
    // Update results
    // TODO: timview mode
    //
    const unsigned int image_index = launch_idx.y * launch_dims.x + launch_idx.x;
    float3             accum_color = payload.result;

    if( subframe_index > 0)
    {
        const float  a                = 1.0f / static_cast<float>( subframe_index + 1 );
        const float3 accum_color_prev = make_float3( whitted::params.accum_buffer[image_index] );
        accum_color                   = lerp( accum_color_prev, accum_color, a );
    }
    whitted::params.accum_buffer[image_index] = make_float4( accum_color, 1.0f );
    whitted::params.frame_buffer[image_index] = make_color( accum_color );
}


extern "C" __global__ void __raygen__pinhole_mdas()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const float3 eye = whitted::params.eye;
    const float3 U = whitted::params.U;
    const float3 V = whitted::params.V;
    const float3 W = whitted::params.W;
    const int    subframe_index = whitted::params.subframe_index;
    const int    linear_index = whitted::params.sample_offset + launch_idx.y * launch_dims.x + launch_idx.x;

    //
    // Generate camera ray
    //
    mdas::Point sample = whitted::params.sample_coordinates[linear_index];
    // The center of each pixel is at fraction (0.5,0.5)
    const float2 d = 2.0f * make_float2(sample[0] / whitted::params.scale.x,  
                                        sample[1] / whitted::params.scale.y) - 1.0f;
    float3 ray_direction = normalize(make_float3(d.x, d.y, 1.0f));
    float3 ray_origin = make_float3(0.0f);

    //
    // Depth of field
    // 
    if (whitted::params.lens_radius > 0.0f)
    {
        // Map uniform random numbers to $[-1,1]^2$
        float2 rnd2 = make_float2(sample[2], sample[3]);
        float2 offset = 2.f * rnd2 - make_float2(1, 1);

        // Handle degeneracy at the origin
        float2 lens = make_float2(0.0f);
        if (offset.x != 0 || offset.y != 0)
        {
            // Apply concentric mapping to point
            float theta, r;
            if (fabs(offset.x) > fabs(offset.y))
            {
                r = offset.x;
                theta = M_PI_4 * (offset.y / offset.x);
            }
            else
            {
                r = offset.y;
                theta = M_PI_2 - M_PI_4 * (offset.x / offset.y);
            }
            lens = whitted::params.lens_radius * r * make_float2(cosf(theta), sinf(theta));
        }

        // Compute point on plane of focus
        float ft = whitted::params.focal_distance / ray_direction.z;
        float3 focus = ft * ray_direction;

        // Update ray for effect of lens
        ray_origin = make_float3(lens.x, lens.y, 0);
        ray_direction = normalize(focus - ray_origin);
    }

    // Transform
    ray_origin += eye;
    ray_direction = normalize(ray_direction.x * U + ray_direction.y * V + ray_direction.z * W);

    //
    // Trace camera ray
    //
    whitted::PayloadRadiance payload;
    payload.result = make_float3(0.0f);
    payload.importance = 1.0f;
    payload.depth = 0.0f;

    traceRadiance(whitted::params.handle, ray_origin, ray_direction,
        0.01f,  // tmin       // TODO: smarter offset
        1e16f,  // tmax
        &payload);

    //
    // Update results
    // TODO: timview mode
    //
    whitted::params.sample_values[linear_index] = payload.result;
}


extern "C" __global__ void __miss__constant_radiance()
{
    whitted::setPayloadResult( whitted::params.miss_color );
}


extern "C" __global__ void __closesthit__occlusion()
{
    whitted::setPayloadOcclusion( true );
}


extern "C" __global__ void __closesthit__radiance()
{
    const whitted::HitGroupData* hit_group_data = reinterpret_cast<whitted::HitGroupData*>( optixGetSbtDataPointer() );
    const LocalGeometry          geom           = getLocalGeometry( hit_group_data->geometry_data );

    //
    // Retrieve material data
    //
    float3 base_color = make_float3( hit_group_data->material_data.pbr.base_color );
    if( hit_group_data->material_data.pbr.base_color_tex )
        base_color *= whitted::linearize(
            make_float3( tex2D<float4>( hit_group_data->material_data.pbr.base_color_tex, geom.UV.x, geom.UV.y ) ) );

    float  metallic  = hit_group_data->material_data.pbr.metallic;
    float  roughness = hit_group_data->material_data.pbr.roughness;
    float4 mr_tex    = make_float4( 1.0f );
    if( hit_group_data->material_data.pbr.metallic_roughness_tex )
        // MR tex is (occlusion, roughness, metallic )
        mr_tex = tex2D<float4>( hit_group_data->material_data.pbr.metallic_roughness_tex, geom.UV.x, geom.UV.y );
    roughness *= mr_tex.y;
    metallic *= mr_tex.z;

    //
    // Convert to material params
    //
    const float  F0         = 0.04f;
    const float3 diff_color = base_color * ( 1.0f - F0 ) * ( 1.0f - metallic );
    const float3 spec_color = lerp( make_float3( F0 ), base_color, metallic );
    const float  alpha      = roughness * roughness;

    //
    // compute direct lighting
    //

    float3 N = geom.N;
    if( hit_group_data->material_data.pbr.normal_tex )
    {
        const float4 NN =
            2.0f * tex2D<float4>( hit_group_data->material_data.pbr.normal_tex, geom.UV.x, geom.UV.y ) - make_float4( 1.0f );
        N = normalize( NN.x * normalize( geom.dpdu ) + NN.y * normalize( geom.dpdv ) + NN.z * geom.N );
    }

    float3 result = make_float3( 0.0f );

    for( int i = 0; i < whitted::params.lights.count; ++i )
    {
        Light light = whitted::params.lights[i];
        if( light.type == Light::Type::POINT )
        {
            // TODO: optimize
            const float  L_dist  = length( light.point.position - geom.P );
            const float3 L       = ( light.point.position - geom.P ) / L_dist;
            const float3 V       = -normalize( optixGetWorldRayDirection() );
            const float3 H       = normalize( L + V );
            const float  N_dot_L = dot( N, L );
            const float  N_dot_V = dot( N, V );
            const float  N_dot_H = dot( N, H );
            const float  V_dot_H = dot( V, H );

            if( N_dot_L > 0.0f && N_dot_V > 0.0f )
            {
                const float tmin     = 0.001f;           // TODO
                const float tmax     = L_dist - 0.001f;  // TODO
                const bool  occluded = whitted::traceOcclusion( whitted::params.handle, geom.P, L, tmin, tmax );
                if( !occluded )
                {
                    const float3 F     = whitted::schlick( spec_color, V_dot_H );
                    const float  G_vis = whitted::vis( N_dot_L, N_dot_V, alpha );
                    const float  D     = whitted::ggxNormal( N_dot_H, alpha );

                    const float3 diff = ( 1.0f - F ) * diff_color / M_PIf;
                    const float3 spec = F * G_vis * D;

                    result += light.point.color * light.point.intensity * N_dot_L * ( diff + spec );
                }
            }
        }
        else if( light.type == Light::Type::AMBIENT )
        {
            result += light.ambient.color * base_color;
        }
    }

    whitted::setPayloadResult( result );
}

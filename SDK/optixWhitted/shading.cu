#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>

#include "optixWhitted.h"
#include "helpers.h"

extern "C" {
__constant__ Params params;
}

static __device__ __inline__ RadiancePRD getRadiancePRD()
{
    RadiancePRD prd;
    prd.result.x = int_as_float( optixGetPayload_0() );
    prd.result.y = int_as_float( optixGetPayload_1() );
    prd.result.z = int_as_float( optixGetPayload_2() );
    prd.importance = int_as_float( optixGetPayload_3() );
    prd.depth = optixGetPayload_4();
    return prd;
}

static __device__ __inline__ void setRadiancePRD( const RadiancePRD &prd )
{
    optixSetPayload_0( float_as_int(prd.result.x) );
    optixSetPayload_1( float_as_int(prd.result.y) );
    optixSetPayload_2( float_as_int(prd.result.z) );
    optixSetPayload_3( float_as_int(prd.importance) );
    optixSetPayload_4( prd.depth );
}

static __device__ __inline__ OcclusionPRD getOcclusionPRD()
{
    OcclusionPRD prd;
    prd.attenuation.x = int_as_float( optixGetPayload_0() );
    prd.attenuation.y = int_as_float( optixGetPayload_1() );
    prd.attenuation.z = int_as_float( optixGetPayload_2() );
    return prd;
}

static __device__ __inline__ void setOcclusionPRD( const OcclusionPRD &prd )
{
    optixSetPayload_0( float_as_int(prd.attenuation.x) );
    optixSetPayload_1( float_as_int(prd.attenuation.y) );
    optixSetPayload_2( float_as_int(prd.attenuation.z) );
}

static __device__ __inline__ float3
traceRadianceRay(
    float3 origin,
    float3 direction,
    int depth,
    float importance)
{
    RadiancePRD prd;
    prd.depth = depth;
    prd.importance = importance;

    optixTrace(
        params.handle,
        origin,
        direction,
        params.scene_epsilon,
        1e16f,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,
        RAY_TYPE_COUNT,
        RAY_TYPE_RADIANCE,
        float3_as_args(prd.result),
        /* Can't use float_as_int() because it returns rvalue but payload requires a lvalue */
        reinterpret_cast<unsigned int&>(prd.importance),
        reinterpret_cast<unsigned int&>(prd.depth) );

    return prd.result;
}

static
__device__ void phongShadowed()
{
    // this material is opaque, so it fully attenuates all shadow rays
    OcclusionPRD prd;
    prd.attenuation = make_float3(0.f);
    setOcclusionPRD(prd);
}

static
__device__ void phongShade( float3 p_Kd,
                            float3 p_Ka,
                            float3 p_Ks,
                            float3 p_Kr,
                            float  p_phong_exp,
                            float3 p_normal )
{
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_t    = optixGetRayTmax();

    RadiancePRD prd = getRadiancePRD();

    float3 hit_point = ray_orig + ray_t * ray_dir;

    // ambient contribution
    float3 result = p_Ka * params.ambient_light_color;

    // compute direct lighting
    BasicLight light = params.light;
    float Ldist = length(light.pos - hit_point);
    float3 L = normalize(light.pos - hit_point);
    float nDl = dot( p_normal, L);

    // cast shadow ray
    float3 light_attenuation = make_float3(static_cast<float>( nDl > 0.0f ));
    if ( nDl > 0.0f )
    {
        OcclusionPRD shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);

        optixTrace(
            params.handle,
            hit_point,
            L,
            0.01f,
            Ldist,
            0.0f,
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_OCCLUSION,
            RAY_TYPE_COUNT,
            RAY_TYPE_OCCLUSION,
            float3_as_args(shadow_prd.attenuation) );

        light_attenuation = shadow_prd.attenuation;
    }

    // If not completely shadowed, light the hit point
    if( fmaxf(light_attenuation) > 0.0f )
    {
        float3 Lc = light.color * light_attenuation;

        result += p_Kd * nDl * Lc;

        float3 H = normalize(L - ray_dir);
        float nDh = dot( p_normal, H );
        if(nDh > 0)
        {
            float power = pow(nDh, p_phong_exp);
            result += p_Ks * power * Lc;
        }
    }

    if( fmaxf( p_Kr ) > 0 )
    {

        // ray tree attenuation
        float new_importance = prd.importance * luminance( p_Kr );
        int new_depth = prd.depth + 1;

        // reflection ray
        // compare new_depth to max_depth - 1 to leave room for a potential shadow ray trace
        if( new_importance >= 0.01f && new_depth <= params.max_depth - 1)
        {
            float3 R = reflect( ray_dir, p_normal );

            result += p_Kr * traceRadianceRay(
                hit_point,
                R,
                new_depth,
                new_importance);
        }
    }

    // pass the color back
    prd.result = result;
    setRadiancePRD(prd);
}

extern "C" __global__ void __closesthit__checker_radiance()
{
    const HitGroupData* sbt_data = (HitGroupData*) optixGetSbtDataPointer();
    const CheckerPhong &checker = sbt_data->shading.checker;

    float3 Kd, Ka, Ks, Kr;
    float  phong_exp;

    float2 texcoord = make_float2(
        int_as_float( optixGetAttribute_3() ),
        int_as_float( optixGetAttribute_4() ) );
    float2 t  = texcoord * checker.inv_checker_size;
    t.x = floorf(t.x);
    t.y = floorf(t.y);

    int which_check = ( static_cast<int>( t.x ) +
                        static_cast<int>( t.y ) ) & 1;

    if ( which_check )
    {
        Kd = checker.Kd1;
        Ka = checker.Ka1;
        Ks = checker.Ks1;
        Kr = checker.Kr1;
        phong_exp = checker.phong_exp1;
    } else
    {
        Kd = checker.Kd2;
        Ka = checker.Ka2;
        Ks = checker.Ks2;
        Kr = checker.Kr2;
        phong_exp = checker.phong_exp2;
    }

    float3 object_normal = make_float3(
        int_as_float( optixGetAttribute_0() ),
        int_as_float( optixGetAttribute_1() ),
        int_as_float( optixGetAttribute_2() ));
    float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace(object_normal) );
    float3 ffnormal  = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );
    phongShade( Kd, Ka, Ks, Kr, phong_exp, ffnormal );
}

extern "C" __global__ void __closesthit__metal_radiance()
{
    const HitGroupData* sbt_data = (HitGroupData*) optixGetSbtDataPointer();
    const Phong &phong = sbt_data->shading.metal;

    float3 object_normal = make_float3(
        int_as_float( optixGetAttribute_0() ),
        int_as_float( optixGetAttribute_1() ),
        int_as_float( optixGetAttribute_2() ));

    float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float3 ffnormal = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );
    phongShade( phong.Kd, phong.Ka, phong.Ks, phong.Kr, phong.phong_exp, ffnormal );
}

extern "C" __global__ void __closesthit__full_occlusion()
{
    phongShadowed();
}

extern "C" __global__ void __closesthit__glass_radiance()
{
    const HitGroupData* sbt_data = (HitGroupData*) optixGetSbtDataPointer();
    const Glass &glass = sbt_data->shading.glass;

    RadiancePRD prd_radiance = getRadiancePRD();

    float3 object_normal = make_float3(
        int_as_float( optixGetAttribute_0() ),
        int_as_float( optixGetAttribute_1() ),
        int_as_float( optixGetAttribute_2() ));
    object_normal = normalize( object_normal );

    // intersection vectors
    const float3 n = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal) ); // normal
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();                 // incident direction
    const float  ray_t    = optixGetRayTmax();
    float3 t;                                                            // transmission direction
    float3 r;                                                            // reflection direction

    float3 hit_point = ray_orig + ray_t * ray_dir;
    SphereShellHitType hit_type = (SphereShellHitType) optixGetHitKind();
    float3 front_hit_point = hit_point, back_hit_point = hit_point;

    if (hit_type & HIT_OUTSIDE_FROM_OUTSIDE || hit_type & HIT_INSIDE_FROM_INSIDE)
    {
        front_hit_point += params.scene_epsilon * object_normal;
        back_hit_point  -= params.scene_epsilon * object_normal;
    }
    else
    {
        front_hit_point -= params.scene_epsilon * object_normal;
        back_hit_point  += params.scene_epsilon * object_normal;
    }

    const float3 fhp = optixTransformPointFromObjectToWorldSpace( front_hit_point );
    const float3 bhp = optixTransformPointFromObjectToWorldSpace( back_hit_point );

    float reflection = 1.0f;
    float3 result = make_float3(0.0f);

    const int depth = prd_radiance.depth;

    float3 beer_attenuation;
    if(dot(n, ray_dir) > 0)
    {
        // Beer's law attenuation
        beer_attenuation = exp(glass.extinction_constant * ray_t);
    } else
    {
        beer_attenuation = make_float3(1);
    }

    // refraction
    // compare depth to max_depth - 1 to leave room for a potential shadow ray trace
    if (depth < min(glass.refraction_maxdepth, params.max_depth - 1))
    {
        if ( refract(t, ray_dir, n, glass.refraction_index) )
        {
            // check for external or internal reflection
            float cos_theta = dot(ray_dir, n);
            if (cos_theta < 0.0f)
                cos_theta = -cos_theta;
            else
                cos_theta = dot(t, n);

            reflection = fresnel_schlick(
                cos_theta,
                glass.fresnel_exponent,
                glass.fresnel_minimum,
                glass.fresnel_maximum);

            float importance =
                prd_radiance.importance
                * (1.0f-reflection)
                * luminance( glass.refraction_color * beer_attenuation );
            float3 color = glass.cutoff_color;
            if ( importance > glass.importance_cutoff )
            {
                color = traceRadianceRay(bhp, t, depth+1, importance);
            }
            result += (1.0f - reflection) * glass.refraction_color * color;
        }
        // else TIR
    } // else reflection==1 so refraction has 0 weight

    // reflection
    // compare depth to max_depth - 1 to leave room for a potential shadow ray trace
    float3 color = glass.cutoff_color;
    if (depth < min(glass.reflection_maxdepth, params.max_depth - 1))
    {
        r = reflect(ray_dir, n);

        float importance =
            prd_radiance.importance
            * reflection
            * luminance( glass.reflection_color * beer_attenuation );
        if ( importance > glass.importance_cutoff )
        {
            color = traceRadianceRay( fhp, r, depth+1, importance );
        }
    }
    result += reflection * glass.reflection_color * color;

    result = result * beer_attenuation;

    prd_radiance.result = result;
    setRadiancePRD(prd_radiance);
}

extern "C" __global__ void __anyhit__glass_occlusion()
{
    const HitGroupData* sbt_data = (HitGroupData*) optixGetSbtDataPointer();
    const Glass &glass = sbt_data->shading.glass;

    float3 object_normal = make_float3(
        int_as_float( optixGetAttribute_0() ),
        int_as_float( optixGetAttribute_1() ),
        int_as_float( optixGetAttribute_2() ));

    OcclusionPRD shadow_prd = getOcclusionPRD();

    float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float nDi = fabs(dot(world_normal, optixGetWorldRayDirection()));

    shadow_prd.attenuation *= 1-fresnel_schlick(nDi, 5, 1-glass.shadow_attenuation, make_float3(1));
    setOcclusionPRD(shadow_prd);

    // Test the attenuation of the light from the glass shell
    if(luminance(shadow_prd.attenuation) < glass.importance_cutoff)
        // The attenuation is so high, > 99% blocked, that we can consider testing to be done.
        optixTerminateRay();
    else
        // There is still some light coming through the glass shell that we should test other occluders.
        // We "ignore" the intersection with the glass shell, meaning that shadow testing will continue.
        // If the ray does not hit another occluder, the light's attenuation from this glass shell
        // (along with other glass shells) is then used.
        optixIgnoreIntersection();
}

extern "C" __global__ void __miss__constant_bg()
{
    const MissData* sbt_data = (MissData*) optixGetSbtDataPointer();
    RadiancePRD prd = getRadiancePRD();
    prd.result = sbt_data->bg_color;
    setRadiancePRD(prd);
}

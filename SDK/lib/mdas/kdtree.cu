#include "hip/hip_runtime.h"
#include <sutil/vec_math.h>
#include <cuda/helpers.h>
#include <cuda/random.h>

#include <stack>
#include <vector>
#include <algorithm>
#include <iostream>
#include "aabb.h"
#include "kdtree.h"

namespace mdas {

#define STACK_SIZE              64          // Size of the traversal stack in local memory.
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

#define divCeil(a, b) (((a) + (b) - 1) / (b))

__device__ int g_warpCounter0;
__device__ int g_warpCounter1;
__device__ float g_error;

texture<float4, 1> t_nodes;
texture<float4, 1> t_nodesxy;
texture<float4, 1> t_nodeszw;

enum {
    MaxBlockHeight = 6,                     // Upper bound for blockDim.y
    EntrypointSentinel = 0x76543210,        // Bottom-most stack entry, indicating the end of traversal
};

float bitsToFloat(int val) {
    return *(float*)&val;
}

int floatToBits(float val) {
    return *(int*)&val;
}

template <class T>
__device__ __inline__ void swap(T& a, T& b) {
    T t = a;
    a = b;
    b = t;
}

__global__ void uniformSamplingKernel(
    int numberOfLeaves,
    int samplesPerLeaf,
    int bitsPerDim,
    int extraImgBits,
    float scaleX,
    float scaleY,
    int* leafIndices,
    Point* sampleCoordinates,
    KDTree::Node* nodes,
    float4* nodesxy,
    float4* nodeszw,
    unsigned int* seeds
) {

    // Leaf index
    const int leafIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (leafIndex < numberOfLeaves) {

        // Cell offset and extent
        Point offset;
        Point extent;
        for (int j = 0; j < Point::DIM; ++j) {
            unsigned int xq = 0;
            unsigned int extentInv = 1 << bitsPerDim;
            for (int k = 0; k < bitsPerDim; ++k) {
                int i = Point::DIM * k + (Point::DIM - j - 1);
                xq |= ((leafIndex >> i) & 1) << k;
            }
            if (j < 2) {
                for (int k = bitsPerDim; k < bitsPerDim + extraImgBits; ++k) {
                    int i = Point::DIM * bitsPerDim + 2 * (k - bitsPerDim) + 1 - j;
                    xq |= ((leafIndex >> i) & 1) << k;
                }
                extentInv <<= extraImgBits;
            }
            extent[j] = 1.0f / float(extentInv);
            offset[j] = xq * extent[j];
        }
        offset[0] *= scaleX;
        offset[1] *= scaleY;
        extent[0] *= scaleX;
        extent[1] *= scaleY;

        // Uniform sampling
        KDTree::Node node;
        unsigned int seed = tea<4>(leafIndex, 0);
        for (int j = 0; j < samplesPerLeaf; ++j) {

            // Random point
            Point r;
            for (int i = 0; i < Point::DIM; ++i) {
                r.data[i] = rnd(seed);
            }

            // Sample index
            int sampleIndex = samplesPerLeaf * leafIndex + j;

            // Transform sample to the cell extent
            sampleCoordinates[sampleIndex] = offset + r * extent;

            // Sample index
            node.indices[j] = sampleIndex;

        }
        seeds[leafIndex] = seed;
        
        // Node index
        int nodeIndex = leafIndex + numberOfLeaves - 1;
        leafIndices[leafIndex] = nodeIndex;

        // Write node
        nodes[nodeIndex] = node;
        nodesxy[nodeIndex] = make_float4(offset[0], offset[0] + extent[0], offset[1], offset[1] + extent[1]);
        nodeszw[nodeIndex] = make_float4(offset[2], offset[2] + extent[2], offset[3], offset[3] + extent[3]);

    }

}

__global__ void constructKernel(
    int numberOfInteriors,
    int maxLeafSize,
    int bitsPerDim,
    int extraImgBits,
    float scaleX,
    float scaleY,
    KDTree::Node* nodes
) {

    // Node index.
    const int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (nodeIndex < numberOfInteriors) {

        // Split dimension
        const int bit = 8 * sizeof(unsigned int) - __clz(unsigned int(nodeIndex + 1)) - 1;
        int dimension = bit < 2 * extraImgBits ? bit & 1 : (bit - 2 * extraImgBits) % Point::DIM;

        // Split position
        unsigned int c = nodeIndex - (1 << bit) + 1;
        float increase = 0.5f;
        float position = 0.0f;
#if 0
        for (int t = bit - dimension - 1; t >= 0; t -= Point::DIM) {
            if ((c >> t) & 1) position += increase;
            increase *= 0.5f;
        }
#else
        int delta = 2;
        int tm = bit - dimension - 1;
        int t0 = tm;
        int th = tm;
        if (dimension < 2) th -= 2 * extraImgBits;
        else t0 -= 2 * extraImgBits;
        for (int t = t0; t >= 0; t -= delta) {
            if ((c >> t) & 1) position += increase;
            if (t <= th) delta = Point::DIM;
            increase *= 0.5f;
        }
#endif
        position += increase;

        // Scale
        if (dimension == 0) position *= scaleX;
        if (dimension == 1) position *= scaleY;

        // Write node
        KDTree::Node node;
        node.right = (nodeIndex + 1) << 1;
        node.left = node.right - 1;
        node.dimension = ~dimension;
        node.position = position;
        nodes[nodeIndex] = node;

    }

}

__global__ void computeBoxesKernel(
    int numberOfNodes,
    float scaleX,
    float scaleY,
    KDTree::Node* nodes,
    float4* nodesxy,
    float4* nodeszw
) {

    // Sample index.
    const int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (nodeIndex < numberOfNodes) {

        // Node
        KDTree::Node node = nodes[0];

        // Bounding box
        AABB box;
        box.mn = Point(0.0f);
        box.mx = Point(1.0f);
        box.mx[0] = scaleX;
        box.mx[1] = scaleY;

        // Split dimension
        const int bit = 8 * sizeof(unsigned int) - __clz(unsigned int(nodeIndex + 1)) - 1;

        // Split position
        unsigned int c = nodeIndex - (1 << bit) + 1;
        for (int t = bit - 1; t >= 0; t--) {
            if ((c >> t) & 1) {
                box.mn[~node.dimension] = node.position;
                node = nodes[node.right];
            }
            else {
                box.mx[~node.dimension] = node.position;
                node = nodes[node.left];
            }
        }

        // Node box
        nodesxy[nodeIndex] = make_float4(box.mn[0], box.mx[0], box.mn[1], box.mx[1]);
        nodeszw[nodeIndex] = make_float4(box.mn[2], box.mx[2], box.mn[3], box.mx[3]);

    }

}

__global__ void updateIndicesKernel(
    int numberOfNodes,
    KDTree::Node* nodes
) {

    // Sample index.
    const int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (nodeIndex < numberOfNodes) {

        // Node
        KDTree::Node node = nodes[nodeIndex];

        // Only interiors
        if (!node.Leaf()) {
            node.left = node.left < 0 ? ~node.left : node.left;
            node.right = node.right < 0 ? ~node.right : node.right;
            if (nodes[node.left].Leaf()) node.left = ~node.left;
            if (nodes[node.right].Leaf()) node.right = ~node.right;
        }

        // Write node
        nodes[nodeIndex] = node;

    }

}

__global__ void computeErrorsKernel(
    int numberOfLeaves,
    int* leafIndices,
    float* errors,
    float3* sampleValues,
    KDTree::Node* nodes,
    float4* nodesxy,
    float4* nodeszw
) {

    // Leaf index.
    const int leafIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (leafIndex < numberOfLeaves) {

        // Node
        int nodeIndex = leafIndices[leafIndex];
        KDTree::Node node = nodes[nodeIndex];

        // Volume
        float4 nodexy = nodesxy[nodeIndex];
        float4 nodezw = nodeszw[nodeIndex];
        float volume = (nodexy.y - nodexy.x) * (nodexy.w - nodexy.z)
            * (nodezw.y - nodezw.x) * (nodezw.w - nodezw.z);

        // Average value
        float3 avgValue = make_float3(0.0f);
        int sampleCount = 0;
        for (int i = 0; i < 4; ++i) {
            if (node.indices[i] >= 0) {
                avgValue += sampleValues[node.indices[i]];
                sampleCount++;
            }
        }
        avgValue /= float(sampleCount);

        // Sum of differences
        float3 diffSum = make_float3(0.0f);
        for (int i = 0; i < sampleCount; ++i) {
            float3 sampleValue = sampleValues[node.indices[i]];
            diffSum.x += fabs(sampleValue.x - avgValue.x);
            diffSum.y += fabs(sampleValue.y - avgValue.y);
            diffSum.z += fabs(sampleValue.z - avgValue.z);
        }

        // Error
        float error = 0.0f;
        if (avgValue.x != 0.0f) error += diffSum.x / avgValue.x;
        if (avgValue.y != 0.0f) error += diffSum.y / avgValue.y;
        if (avgValue.z != 0.0f) error += diffSum.z / avgValue.z;
        error /= float(sampleCount);
        error += 1.0e-5;
        error *= volume;

        // Write error
        errors[leafIndex] = error;
        
        // Max error (prefix scan?)
        if (error > g_error)
            atomicMax((int*)&g_error, __float_as_int(error));

    }

}

__global__ void adaptiveSamplingKernel(
    int numberOfLeaves,
    int numberOfSamples,
    int candidatesNum,
    float errorThreshold,
    float scaleX,
    float scaleY,
    int* outNodeIndices,
    int* leafIndices,
    float* errors,
    unsigned long long* nodeLocks,
    KDTree::Node* nodes,
    float4* nodesxy,
    float4* nodeszw,
    Point* leafSamples,
    Point* sampleCoordinates,
    unsigned int* seeds
) {

    // Leaf index.
    const int leafIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (leafIndex < numberOfLeaves) {

        // Error
        float error = errors[leafIndex];

        if (error >= errorThreshold * g_error) {

            // Node index
            int nodeIndex = leafIndices[leafIndex];

            // Box
            AABB box;
            float4 nodexy = nodesxy[nodeIndex];
            float4 nodezw = nodeszw[nodeIndex];
            box.mn[0] = nodexy.x;
            box.mn[1] = nodexy.z;
            box.mn[2] = nodezw.x;
            box.mn[3] = nodezw.z;
            box.mx[0] = nodexy.y;
            box.mx[1] = nodexy.w;
            box.mx[2] = nodezw.y;
            box.mx[3] = nodezw.w;

            // Best candidate method
            int outNodeIndex = -1;
            float maxDistance = -1.0;
            Point maxCandidate;
            Point center = box.Center();
            unsigned int seed = seeds[leafIndex];
            if (seed == 0) seed = tea<4>(leafIndex, 0);
            for (int j = 0; j < candidatesNum; ++j) {

                // Generate candidate
                Point candidate;

                while (true) {

                    // Sample point bounding sphere
                    Point direction;
                    do {
                        Point r;
                        for (int i = 0; i < Point::DIM; ++i) 
                            r.data[i] = rnd(seed);
                        direction = 2.0f * r - 1.0f;
                    } while (Point::Norm(direction) > 1.0f);
                    const float R = 0.55f;
                    float radius = R * Point::Distance(box.mx, box.mn);
                    candidate = center + radius * direction;

                    // Check extent
                    bool valid = true;
                    for (int i = 2; i < Point::DIM; ++i) {
                        if (candidate[i] < 0.0f || candidate[i] >= 1.0f) {
                            valid = false;
                            break;
                        }
                    }
                    if (candidate[0] < 0.0f || candidate[0] >= scaleX) valid = false;
                    if (candidate[1] < 0.0f || candidate[1] >= scaleY) valid = false;
                    if (valid) break;

                }

                // Nearest neighbor (simplified)
                int curNodeIndex = 0;
                KDTree::Node curNode = nodes[curNodeIndex];
                while (!curNode.Leaf()) {
                    if (candidate[~curNode.dimension] < curNode.position)
                        curNodeIndex = curNode.left < 0 ? ~curNode.left : curNode.left;
                    else
                        curNodeIndex = curNode.right < 0 ? ~curNode.right : curNode.right;
                    curNode = nodes[curNodeIndex];
                }

                // Test samples in the leaf
                float minDistance = FLT_MAX;
                for (int i = 0; i < 4; ++i) {
                    if (curNode.indices[i] >= 0) {
                        float distance = Point::Distance(candidate, sampleCoordinates[curNode.indices[i]]);
                        if (minDistance > distance) {
                            minDistance = distance;
                        }
                    }
                }

                // Distance to the nearest neighbor
                if (maxDistance < minDistance) {
                    maxDistance = minDistance;
                    maxCandidate = candidate;
                    outNodeIndex = curNodeIndex;
                }

            }
            seeds[leafIndex] = seed;

            // Sample coordinates and node index
            leafSamples[leafIndex] = maxCandidate;
            outNodeIndices[leafIndex] = outNodeIndex;

            // Lock node
            unsigned long long lock = (unsigned long long(__float_as_int(error)) << 32ull) | unsigned long long(leafIndex);
            atomicMax(&nodeLocks[outNodeIndex], lock);

        }

    }

}

__global__ void splitKernel(
    int numberOfLeaves,
    int numberOfNodes,
    int numberOfSamples,
    int maxLeafSize,
    int* outNodeIndices,
    unsigned long long* nodeLocks,
    float* errors,
    KDTree::Node* nodes,
    float4* nodesxy,
    float4* nodeszw,
    Point* leafSamples,
    Point* sampleCoordinates
) {

    // Leaf index.
    const int leafIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Warp thread index.
    const int warpThreadIndex = threadIdx.x & 31;

    // Sample indices
    int sampleIndicesLoc[5];

    if (leafIndex < numberOfLeaves) {

        // Error
        float error = errors[leafIndex];

        // Out node index
        int outNodeIndex = outNodeIndices[leafIndex];

        // Lock node
        unsigned long long lock = (unsigned long long(__float_as_int(error)) << 32ull) | unsigned long long(leafIndex);

        // Node was successfuly locked
        if (nodeLocks[outNodeIndex] == lock) {
            
            // Sample index
            int sampleIndex;
            {
                // Prefix scan
                const unsigned int activeMask = __activemask();
                const int warpCount = __popc(activeMask);
                const int warpIndex = __popc(activeMask & ((1u << warpThreadIndex) - 1));
                const int warpLeader = __ffs(activeMask) - 1;

                // Atomically add to global counter and exchange the offset
                int warpOffset;
                if (warpThreadIndex == warpLeader)
                    warpOffset = atomicAdd(&g_warpCounter0, warpCount);
                warpOffset = __shfl_sync(activeMask, warpOffset, warpLeader);
                sampleIndex = numberOfSamples + warpOffset + warpIndex;
            }

            // Sample coordinates
            sampleCoordinates[sampleIndex] = leafSamples[leafIndex];
            
            // Node
            KDTree::Node node = nodes[outNodeIndex];

            // Sample indices
            int sampleCount = 0;
            for (int i = 0; i < 4; ++i) {
                if (node.indices[i] >= 0) {
                    sampleIndicesLoc[i] = node.indices[i];
                    sampleCount++;
                }
            }

            // Enough space for new sample => Just insert sample index
            if (sampleCount < maxLeafSize) {
                node.indices[sampleCount] = sampleIndex;
            }

            // Leaf is full => Split
            else {

                // Box
                AABB box;
                float4 nodexy = nodesxy[outNodeIndex];
                float4 nodezw = nodeszw[outNodeIndex];
                box.mn[0] = nodexy.x;
                box.mn[1] = nodexy.z;
                box.mn[2] = nodezw.x;
                box.mn[3] = nodezw.z;
                box.mx[0] = nodexy.y;
                box.mx[1] = nodexy.w;
                box.mx[2] = nodezw.y;
                box.mx[3] = nodezw.w;

                // New sample index
                sampleIndicesLoc[sampleCount++] = sampleIndex;

                // Split dimension
                int splitDimension = box.LargestAxis();
                node.dimension = ~splitDimension;

                // Sort 
                for (int i = 0; i < sampleCount - 1; i++) {
                    for (int j = 0; j < sampleCount - i - 1; j++) {
                        int a = sampleIndicesLoc[j];
                        int b = sampleIndicesLoc[j + 1];
                        if (sampleCoordinates[a][splitDimension] > sampleCoordinates[b][splitDimension]) {
                            swap(sampleIndicesLoc[j], sampleIndicesLoc[j + 1]);
                        }
                    }
                }

                // Split position
                int md = sampleCount / 2;
                float splitPosition = 0.5f * (sampleCoordinates[sampleIndicesLoc[md - 1]][splitDimension] +
                    sampleCoordinates[sampleIndicesLoc[md]][splitDimension]);
                node.position = splitPosition;

                // Node offset
                int nodeOffset;
                {
                    // Prefix scan
                    const unsigned int activeMask = __activemask();
                    const int warpCount = __popc(activeMask);
                    const int warpIndex = __popc(activeMask & ((1u << warpThreadIndex) - 1));
                    const int warpLeader = __ffs(activeMask) - 1;

                    // Atomically add to global counter and exchange the offset
                    int warpOffset;
                    if (warpThreadIndex == warpLeader)
                        warpOffset = atomicAdd(&g_warpCounter1, warpCount);
                    warpOffset = __shfl_sync(activeMask, warpOffset, warpLeader);
                    nodeOffset = numberOfNodes + 2 * (warpOffset + warpIndex);
                }

                // Chil indices
                node.left = nodeOffset;
                node.right = nodeOffset + 1;

                // Left child
                KDTree::Node left;
                for (int i = 0; i < 4; ++i) {
                    if (i < md) left.indices[i] = sampleIndicesLoc[i];
                    else left.indices[i] = ~Point::DIM;
                }
                nodes[node.left] = left;

                // Left box
                AABB leftBox = box;
                leftBox.mx[splitDimension] = splitPosition;
                nodesxy[node.left] = make_float4(leftBox.mn[0], leftBox.mx[0], leftBox.mn[1], leftBox.mx[1]);
                nodeszw[node.left] = make_float4(leftBox.mn[2], leftBox.mx[2], leftBox.mn[3], leftBox.mx[3]);

                // Right child
                KDTree::Node right;
                for (int i = 0; i < 4; ++i) {
                    if (i < sampleCount - md) right.indices[i] = sampleIndicesLoc[md + i];
                    else right.indices[i] = ~Point::DIM;
                }
                nodes[node.right] = right;

                // Right box
                AABB rightBox = box;
                rightBox.mn[splitDimension] = splitPosition;
                nodesxy[node.right] = make_float4(rightBox.mn[0], rightBox.mx[0], rightBox.mn[1], rightBox.mx[1]);
                nodeszw[node.right] = make_float4(rightBox.mn[2], rightBox.mx[2], rightBox.mn[3], rightBox.mx[3]);

            }

            // Write node
            nodes[outNodeIndex] = node;

        }

    }

}

__global__ void prepareLeafIndicesKernel(
    int numberOfLeaves,
    int* leafIndices0,
    int* leafIndices1,
    KDTree::Node* nodes
) {

    // Leaf index.
    const int leafIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Warp thread index.
    const int warpThreadIndex = threadIdx.x & 31;

    if (leafIndex < numberOfLeaves) {

        // Node
        int nodeIndex = leafIndices0[leafIndex];
        KDTree::Node node = nodes[nodeIndex];

        // Prefix scan
        const unsigned int activeMask = __activemask();
        const unsigned int warpBallot = __ballot_sync(activeMask, !node.Leaf());
        const int warpThreads = __popc(activeMask);
        const int warpCount = __popc(warpBallot);
        const int warpIndex = __popc(warpBallot & ((1u << warpThreadIndex) - 1));

        // Not splitted => Just copy leaf index
        if (node.Leaf()) {

            // Atomically add to global counter and exchange the offset
            int warpOffset;
            const unsigned int activeMaskLeaf = __activemask();
            const int warpLeader = __ffs(activeMaskLeaf) - 1;
            if (warpThreadIndex == warpLeader)
                warpOffset = atomicAdd(&g_warpCounter1, warpThreads - warpCount);
            warpOffset = __shfl_sync(activeMaskLeaf, warpOffset, warpLeader);

            // Leaf index
            int newLeafIndex = warpOffset + (warpThreadIndex - warpIndex);
            leafIndices1[newLeafIndex] = nodeIndex;
            
        }

        // Split => Place new child indices
        else {

            // Atomically add to global counter and exchange the offset
            int warpOffset;
            const unsigned int activeMaskInterior = __activemask();
            const int warpLeader = __ffs(activeMaskInterior) - 1;
            if (warpThreadIndex == warpLeader)
                warpOffset = atomicAdd(&g_warpCounter0, warpCount);
            warpOffset = __shfl_sync(activeMaskInterior, warpOffset, warpLeader);

            // New leaf indices
            leafIndices1[numberOfLeaves - 1 - (warpOffset + warpIndex)] = node.left;
            leafIndices1[numberOfLeaves + warpOffset + warpIndex] = node.right;

        }

    }

}

__global__ void integrateKernel(
    int width,
    int height,
    float scaleX,
    float scaleY,
    float3* sampleValues,
    float4* pixels,
    uchar4* pixelsBytes
) {

    // Traversal stack in CUDA thread-local memory
    int traversalStack[STACK_SIZE];
    traversalStack[0] = EntrypointSentinel; // Bottom-most entry

    // Live state during traversal, stored in registers
    char*   stackPtr;                       // Current position in traversal stack
    int     leafAddr;                       // First postponed leaf, non-negative if none
    int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf
    int     pixIdx;
    int     pixX, pixY;
    float   pMinX, pMaxX, pMinY, pMaxY;
    float   pixArea = (scaleX * scaleY) / (width * height);
    float3  value;

    // Initialize persistent threads.
    __shared__ volatile int nextPixArray[MaxBlockHeight]; // Current ray index in global buffer

    // Persistent threads: fetch and process rays in a loop
    do {
        const int tidx = threadIdx.x;
        volatile int& pixBase = nextPixArray[threadIdx.y];

        // Fetch new rays from the global pool using lane 0
        const bool          terminated = nodeAddr == EntrypointSentinel;
        //const unsigned int  maskTerminated = __ballot_sync(0xffffffff, terminated);
        const unsigned int  maskTerminated = __ballot_sync(__activemask(), terminated);
        const int           numTerminated = __popc(maskTerminated);
        const int           idxTerminated = __popc(maskTerminated & ((1u << tidx) - 1));

        if (terminated) {

            if (idxTerminated == 0)
                pixBase = atomicAdd(&g_warpCounter0, numTerminated);

            pixIdx = pixBase + idxTerminated;
            if (pixIdx >= width * height)
                break;

            // Value
            value = make_float3(0.0f);

            // Compute pixel
            pixX = pixIdx % width;
            pixY = pixIdx / width;
            pMinX = pixX / float(width) * scaleX;
            pMaxX = (pixX + 1) / float(width) * scaleX;
            pMinY = pixY / float(height) * scaleY;
            pMaxY = (pixY + 1) / float(height) * scaleY;

            // Setup traversal
            stackPtr = (char*)&traversalStack[0];
            leafAddr = 0;   // No postponed leaf
            nodeAddr = 0;   // Start from the root
        }

        // Traversal loop
        while (nodeAddr != EntrypointSentinel) {
            // Traverse internal nodes until all SIMD lanes have found a leaf
            while (unsigned int(nodeAddr) < unsigned int(EntrypointSentinel)) {

                // Fetch AABBs of the two child nodes
                float4 tmp = tex1Dfetch(t_nodes, nodeAddr); // child_index0, child_index1
                const KDTree::Node  node = *(KDTree::Node*)&tmp;

                // Intersect the pixel volume with the child nodes
                bool traverseChild0x = ~node.dimension == 0 ? pMinX < node.position : true;
                bool traverseChild0y = ~node.dimension == 1 ? pMinY < node.position : true;
                bool traverseChild1x = ~node.dimension == 0 ? pMaxX > node.position : true;
                bool traverseChild1y = ~node.dimension == 1 ? pMaxY > node.position : true;

                bool traverseChild0 = traverseChild0x && traverseChild0y;
                bool traverseChild1 = traverseChild1x && traverseChild1y;

                // Neither child was intersected => pop stack
                if (!traverseChild0 && !traverseChild1) {
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // Otherwise => fetch child pointers
                else {
                    nodeAddr = (traverseChild0) ? node.left : node.right;

                    // Both children were intersected => push the farther one
                    if (traverseChild0 && traverseChild1) {
                        stackPtr += 4;
                        *(int*)stackPtr = node.right;
                    }
                }

                // First leaf => postpone and continue traversal
                if (nodeAddr < 0 && leafAddr >= 0) {
                    leafAddr = nodeAddr;
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // All SIMD lanes have found a leaf? => process them

                // NOTE: inline PTX implementation of "if(!__any(leafAddr >= 0)) break;"
                // tried everything with CUDA 4.2 but always got several redundant instructions

                //unsigned int mask;
                //asm("{\n"
                //    "   .reg .pred p;               \n"
                //    "setp.ge.s32        p, %1, 0;   \n"
                //    "vote.ballot.b32    %0,p;       \n"
                //    "}"
                //    : "=r"(mask)
                //    : "r"(leafAddr));
                //if (!mask)
                //    break;

                if(!__any_sync(__activemask(), leafAddr >= 0))
                    break;

            }

            // Process postponed leaf nodes
            while (leafAddr < 0) {

                // Node
                float4 tmp = tex1Dfetch(t_nodes, ~leafAddr);
                const KDTree::Node leaf = *(KDTree::Node*)&tmp;

                // Average value
                float3 sampleValue = make_float3(0.0f);
                int sampleCount = 0;
                for (int i = 0; i < 4; ++i) {
                    if (leaf.indices[i] >= 0) {
                        sampleValue += sampleValues[leaf.indices[i]];
                        sampleCount++;
                    }
                }
                sampleValue /= float(sampleCount);

                // Leaf box
                const float4 nxy = tex1Dfetch(t_nodesxy, ~leafAddr); // (c.lo.x, c.hi.x, c.lo.z, c.hi.z)
                const float4 nzw = tex1Dfetch(t_nodeszw, ~leafAddr); // (c.lo.z, c.hi.z, c.lo.w, c.hi.w)

                // Intersect the pixel volume with the leaf
                const float clox = fmax(pMinX, nxy.x);
                const float chix = fmin(pMaxX, nxy.y);
                const float cloy = fmax(pMinY, nxy.z);
                const float chiy = fmin(pMaxY, nxy.w);

                // Volume
                float volume = (chix - clox) * (chiy - cloy) * (nzw.y - nzw.x) * (nzw.w - nzw.z);

                // Add contribution
                value += sampleValue * volume / pixArea;

                // Another leaf was postponed => process it as well.
                leafAddr = nodeAddr;
                if (nodeAddr < 0) {
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }
            } // leaf

            // DYNAMIC FETCH
            if (__popc(__activemask()) < DYNAMIC_FETCH_THRESHOLD)
                break;

        } // traversal

        // Store the result
        pixels[pixIdx] = make_float4(value, 1.0f);
        pixelsBytes[pixIdx] = make_color(value);

    } while (true);

}

KDTree::KDTree(int maxSamples, const std::string& logFilename) : 
    maxLeafSize(4), 
    candidatesNum(1), 
    bitsPerDim(0), 
    extraImgBits(7), 
    numberOfSamples(0), 
    numberOfNodes(0), 
    maxSamples(maxSamples), 
    scaleX(1024.0f), 
    scaleY(512.0f), 
    errorThreshold(0.1f)
{
    seeds.Resize(maxSamples);
    sampleCoordinates.Resize(maxSamples);
    sampleValues.Resize(maxSamples);
    nodes.Resize(2 * maxSamples - 1);
    nodesxy.Resize(2 * maxSamples - 1);
    nodeszw.Resize(2 * maxSamples - 1);
    nodeLocks.Resize(2 * maxSamples - 1);
    outNodeIndices.Resize(maxSamples);
    leafSamples.Resize(maxSamples);
    leafIndices[0].Resize(maxSamples);
    leafIndices[1].Resize(maxSamples);
    errors.Resize(maxSamples);
    int numberOfLeaves = (1 << (bitsPerDim * Point::DIM)) << (extraImgBits << 1);
    int numberOfInitialSamples = numberOfLeaves * maxLeafSize;
    std::cout << "Initial samples " << numberOfInitialSamples << std::endl;
    if (!logFilename.empty()) {
        std::cout << logFilename << std::endl;
        logStats = true;
        log.open(logFilename + ".log");
    }
}

KDTree::~KDTree() {
    if (logStats) log.close();
}

void KDTree::InitialSampling() {

    // Reset seeds
    hipMemset(seeds.Data(), 0, sizeof(unsigned int) * maxSamples);

    // Number of samples
    int numberOfLeaves = (1 << (bitsPerDim  * Point::DIM)) << (extraImgBits << 1);
    numberOfSamples = numberOfLeaves * maxLeafSize;

    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        uniformSamplingKernel, 0, 0);
    int gridSize = divCeil(numberOfLeaves, blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    uniformSamplingKernel<<<gridSize, blockSize>>>(numberOfLeaves, maxLeafSize, bitsPerDim, extraImgBits, scaleX, scaleY,
        leafIndices[0].Data(), sampleCoordinates.Data(), nodes.Data(), nodesxy.Data(), nodeszw.Data(), seeds.Data());

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "INITIAL SAMPLING\n" << time << std::endl;
    }

}

void KDTree::Construct(void) {

    // Number of nodes
    int numberOfLeaves = numberOfSamples / maxLeafSize;
    numberOfNodes = 2 * numberOfLeaves - 1;

    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        constructKernel, 0, 0);
    int gridSize = divCeil(numberOfLeaves - 1, blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    constructKernel<<<gridSize, blockSize>>>(numberOfLeaves - 1, maxLeafSize, bitsPerDim,
        extraImgBits, scaleX, scaleY, nodes.Data());

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "CONSTRUCT\n" << time << std::endl;
    }

}

void KDTree::UpdateIndices(void) {
    
    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        updateIndicesKernel, 0, 0);
    int gridSize = divCeil(numberOfNodes, blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    updateIndicesKernel<<<gridSize, blockSize>>>(numberOfNodes, nodes.Data());

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "UPDATE INDICES\n" << time << std::endl;
    }

}

void KDTree::ComputeErrors(void) {

    // Reset atomic counter
    const float zero = 0.0f;
    hipMemcpyToSymbol(HIP_SYMBOL(g_error), &zero, sizeof(float));

    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        computeErrorsKernel, 0, 0);
    int gridSize = divCeil(GetNumberOfLeaves(), blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    computeErrorsKernel<<<gridSize, blockSize>>>(GetNumberOfLeaves(), leafIndices[swapBuffers].Data(), 
        errors.Data(), sampleValues.Data(), nodes.Data(), nodesxy.Data(), nodeszw.Data());

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "COMPUTE ERRORS\n" << time << std::endl;
    }

}

void KDTree::AdaptiveSampling(void) {

    // Reset locks
    hipMemset(nodeLocks.Data(), 0, sizeof(unsigned long long) * GetNumberOfNodes());

    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        adaptiveSamplingKernel, 0, 0);
    int gridSize = divCeil(GetNumberOfLeaves(), blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    adaptiveSamplingKernel<<<gridSize, blockSize>>>(
        GetNumberOfLeaves(), 
        numberOfSamples, 
        candidatesNum, 
        errorThreshold, 
        scaleX,
        scaleY,
        outNodeIndices.Data(), 
        leafIndices[swapBuffers].Data(), 
        errors.Data(), 
        nodeLocks.Data(), 
        nodes.Data(),
        nodesxy.Data(), 
        nodeszw.Data(), 
        leafSamples.Data(),
        sampleCoordinates.Data(),
        seeds.Data()
    );

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "ADAPTIVE SAMPLING\n" << time << std::endl;
    }

}

void KDTree::Split(void) {

    // Reset atomic counter
    const int zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter0), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter1), &zero, sizeof(int));
    
    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        splitKernel, 0, 0);
    int gridSize = divCeil(GetNumberOfLeaves(), blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    splitKernel<<<gridSize, blockSize>>>(
        GetNumberOfLeaves(),
        numberOfNodes,
        numberOfSamples,
        maxLeafSize,
        outNodeIndices.Data(),
        nodeLocks.Data(),
        errors.Data(),
        nodes.Data(),
        nodesxy.Data(),
        nodeszw.Data(),
        leafSamples.Data(),
        sampleCoordinates.Data()
    );

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "SPLIT\n" << time << std::endl;
    }

    // Number of samples
    hipMemcpyFromSymbol(&newSamples, HIP_SYMBOL(g_warpCounter0), sizeof(int), 0);
    numberOfSamples += newSamples;

}

void KDTree::PrepareLeafIndices(void) {

    // Reset atomic counter
    const int zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter0), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter1), &zero, sizeof(int));

    // Grid and block size
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
        prepareLeafIndicesKernel, 0, 0);
    int gridSize = divCeil(GetNumberOfLeaves(), blockSize);

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    prepareLeafIndicesKernel<<<gridSize, blockSize>>>(
        GetNumberOfLeaves(),
        leafIndices[swapBuffers].Data(),
        leafIndices[!swapBuffers].Data(),
        nodes.Data()
     );

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "PREPARE LEAF INDICES\n" << time << std::endl;
    }

    // Number of nodes
    int newInteriors;
    hipMemcpyFromSymbol(&newInteriors, HIP_SYMBOL(g_warpCounter0), sizeof(int), 0);

    // Number of nodes
    int oldNumberOfLeaves = GetNumberOfLeaves();
    numberOfNodes += 2 * newInteriors;

    // Check counts
    if (oldNumberOfLeaves + newInteriors != GetNumberOfLeaves()) {
        std::cout << "Number of leaves is not consistent! " << oldNumberOfLeaves
            + newInteriors << " != " << GetNumberOfLeaves() << std::endl;
    }

}

void KDTree::Build() {
    InitialSampling();
    Construct();
    swapBuffers = false;
}

void KDTree::SamplingPass(void) {
    ComputeErrors();
    AdaptiveSampling();
    Split();
    PrepareLeafIndices();
    swapBuffers = !swapBuffers;
}

void KDTree::Integrate(float4* pixels, uchar4* pixelsBytes, int width, int height) {

    // Grid and block size
    const int desiredWarps = 720;
    dim3 blockSize(32, 4);
    int blockWarps = (blockSize.x * blockSize.y + 31) / 32; // 4
    int gridSize = (desiredWarps + blockWarps - 1) / blockWarps;

    // Setup texture references
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    CUDA_CHECK(hipBindTexture(0, &t_nodes, nodes.Data(), &desc, sizeof(float4) * nodes.Size()));
    CUDA_CHECK(hipBindTexture(0, &t_nodesxy, nodesxy.Data(), &desc, sizeof(float4) * nodesxy.Size()));
    CUDA_CHECK(hipBindTexture(0, &t_nodeszw, nodeszw.Data(), &desc, sizeof(float4) * nodeszw.Size()));

    // Reset atomic counter
    const int zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter0), &zero, sizeof(int));

    // Timer
    hipEvent_t start, stop;
    if (logStats) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }

    // Launch
    integrateKernel<<<gridSize, blockSize>>>(width, height, scaleX, scaleY, sampleValues.Data(), pixels, pixelsBytes);

    // Elapsed time and cleanup
    if (logStats) {
        float time;
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        log << "INTEGRATE\n" << time << std::endl;
    }

}

void KDTree::SamplingDensity(float4* pixels, int width, int height) {
    float samplingDensity = 0.05f;
    memset(pixels, 0, sizeof(float4) * width * height);
    for (int i = 0; i < GetNumberOfSamples(); ++i) {
        int x = sampleCoordinates[i][0] / scaleX * width;
        int y = sampleCoordinates[i][1] / scaleY * height;
        pixels[y * width + x] += make_float4(samplingDensity);
    }
    for (int i = 0; i < width * height; ++i)
        pixels[i].w = 1.0f;
}

bool KDTree::Validate(void) {

    // Volume, value, and number of samples
    float totalVolume = 0.0f;
    float3 totalValue = make_float3(0.0f);
    int totalSampleCount = 0;

    // Stack
    std::stack<int> stack;
    stack.push(0);

    // Sample index histogram
    std::vector<int> sampleHist(numberOfSamples);
    memset(sampleHist.data(), 0, sizeof(int) * numberOfSamples);

    // Validation flag
    bool valid = true;

    // Find leaves and validate them
    while (!stack.empty()) {

        // Pop node index
        int nodeIndex = stack.top();
        stack.pop();

        // Box
        AABB box;
        box.mn.data[0] = nodesxy[nodeIndex].x;
        box.mn.data[1] = nodesxy[nodeIndex].z;
        box.mn.data[2] = nodeszw[nodeIndex].x;
        box.mn.data[3] = nodeszw[nodeIndex].z;
        box.mx.data[0] = nodesxy[nodeIndex].y;
        box.mx.data[1] = nodesxy[nodeIndex].w;
        box.mx.data[2] = nodeszw[nodeIndex].y;
        box.mx.data[3] = nodeszw[nodeIndex].w;

        // Leaf
        if (nodes[nodeIndex].Leaf()) {

            float3 avgValue = make_float3(0.0f);
            int sampleCount = 0;
            for (int i = 0; i < 4; ++i) {
                if (nodes[nodeIndex].indices[i] >= 0) {
                    int sampleIndex = nodes[nodeIndex].indices[i];
                    if (!box.Contains(sampleCoordinates[sampleIndex])) {
                        valid = false;
                        std::cout << "Sample is outside the leaf!" << std::endl;
                        std::cout << "Box min " << box.mn[0] << " " << box.mn[1] << " " << box.mn[2] << " " << box.mn[3] << std::endl;
                        std::cout << "Sample  " << sampleCoordinates[sampleIndex][0] << " " << sampleCoordinates[sampleIndex][1] << " " 
                            << sampleCoordinates[sampleIndex][2] << " " << sampleCoordinates[sampleIndex][3] << std::endl;
                        std::cout << "Box max " << box.mx[0] << " " << box.mx[1] << " " << box.mx[2] << " " << box.mx[3] << std::endl;
                    }
                    avgValue += sampleValues[sampleIndex];
                    ++sampleHist[sampleIndex];
                    ++sampleCount;
                }
            }
            avgValue /= float(sampleCount);

            // Add volume and sample count
            totalVolume += box.Volume();
            totalSampleCount += sampleCount;
            totalValue += avgValue * box.Volume();

        }

        // Interior
        else {
            int rightIndex = nodes[nodeIndex].right < 0 ? ~nodes[nodeIndex].right : nodes[nodeIndex].right;
            int leftIndex = nodes[nodeIndex].left < 0 ? ~nodes[nodeIndex].left : nodes[nodeIndex].left;
            stack.push(rightIndex);
            stack.push(leftIndex);
        }

    }

    float rootVolume = scaleX * scaleY;
    if (abs(totalVolume - rootVolume) > 1.0e-2 * rootVolume) {
        std::cout << "Total volume bounded by leaves is not equal to the volume of bounded by the root " <<
            totalVolume << " != " << rootVolume << std::endl;
        valid = false;
    }
    if (totalSampleCount != numberOfSamples) {
        std::cout << "Number of samples is different than number of indices in leaves " <<
            numberOfSamples << " != " << totalSampleCount << std::endl;
        valid = false;
    }
    for (int i = 0; i < numberOfSamples; ++i) {
        if (sampleHist[i] != 1) {
            valid = false;
            std::cout << "Sample not referenced or referenced more than once "  << i << " " << sampleHist[i] << ": ";
            std::cout << sampleCoordinates[i][0] << " " << sampleCoordinates[i][1] 
               << " " << sampleCoordinates[i][2] << " " << sampleCoordinates[i][3] << std::endl;
            for (int k = 0; k < GetNumberOfLeaves(); ++k) {
                for (int j = 0; j < 4; ++j) {
                    int nodeIndex = leafIndices[swapBuffers][k];
                    KDTree::Node curNode = nodes[nodeIndex];
                    if (curNode.indices[j] == i) {
                        AABB box;
                        box.mn.data[0] = nodesxy[nodeIndex].x;
                        box.mn.data[1] = nodesxy[nodeIndex].z;
                        box.mn.data[2] = nodeszw[nodeIndex].x;
                        box.mn.data[3] = nodeszw[nodeIndex].z;
                        box.mx.data[0] = nodesxy[nodeIndex].y;
                        box.mx.data[1] = nodesxy[nodeIndex].w;
                        box.mx.data[2] = nodeszw[nodeIndex].y;
                        box.mx.data[3] = nodeszw[nodeIndex].w;
                        std::cout << "Sample is in leaf node " << nodeIndex << std::endl;
                        std::cout << "\t" << box.mn[0] << " " << box.mn[1] << " " << box.mn[2] << " " << box.mn[3] << std::endl;
                        std::cout << "\t" << box.mx[0] << " " << box.mx[1] << " " << box.mx[2] << " " << box.mx[3] << std::endl;
                    }
                }
            }
            //Point candidate = sampleCoordinates[i];
            //int curNodeIndex = 0;
            //KDTree::Node curNode = nodes[curNodeIndex];
            //while (!curNode.Leaf()) {
            //    if (candidate[~curNode.dimension] < curNode.position)
            //        curNodeIndex = curNode.left < 0 ? ~curNode.left : curNode.left;
            //    else
            //        curNodeIndex = curNode.right < 0 ? ~curNode.right : curNode.right;
            //    curNode = nodes[curNodeIndex];
            //}
            //std::cout << "indices:\n";
            //for (int j = 0; j < 4; ++j)
            //    std::cout << curNode.indices[i] << " ";
            //std::cout << "done" << std::endl;
        }
    }

    // Test traversal (splitting planes)
    for (int i = 0; i < numberOfSamples; ++i) {

        // Find leaf
        stack.push(0);
        Point sample = sampleCoordinates[i];
        bool contains = false;
        while (!stack.empty()) {
            int curNodeIndex = stack.top();
            KDTree::Node curNode = nodes[curNodeIndex];
            stack.pop();
            if (curNode.Leaf()) {
                for (int j = 0; j < 4; ++j) {
                    if (curNode.indices[j] == i)
                        contains = true;
                }
            }
            else {
                if (sample[~curNode.dimension] <= curNode.position)
                    stack.push(curNode.left < 0 ? ~curNode.left : curNode.left);
                if (sample[~curNode.dimension] >= curNode.position)
                    stack.push(curNode.right < 0 ? ~curNode.right : curNode.right);
            }
        }

        if (!contains) {
            valid = false;
            std::cout << "Sample is not in any leaf " << i << ": ";
            std::cout << sampleCoordinates[i][0] << " " << sampleCoordinates[i][1] << 
                  " " << sampleCoordinates[i][2] << " " << sampleCoordinates[i][3] << std::endl;
            std::cout << "Histogram " << sampleHist[i] << std::endl;
            for (int k = 0; k < GetNumberOfLeaves(); ++k) {
                for (int j = 0; j < 4; ++j) {
                    int nodeIndex = leafIndices[swapBuffers][k];
                    KDTree::Node curNode = nodes[nodeIndex];
                    if (curNode.indices[j] == i) {
                        contains = true;
                        AABB box;
                        box.mn.data[0] = nodesxy[nodeIndex].x;
                        box.mn.data[1] = nodesxy[nodeIndex].z;
                        box.mn.data[2] = nodeszw[nodeIndex].x;
                        box.mn.data[3] = nodeszw[nodeIndex].z;
                        box.mx.data[0] = nodesxy[nodeIndex].y;
                        box.mx.data[1] = nodesxy[nodeIndex].w;
                        box.mx.data[2] = nodeszw[nodeIndex].y;
                        box.mx.data[3] = nodeszw[nodeIndex].w;
                        std::cout << "Sample is in leaf node " << nodeIndex << std::endl;
                        std::cout << "\t" << box.mn[0] << " " << box.mn[1] << " " << box.mn[2] << " " << box.mn[3] << std::endl;
                        std::cout << "\t" << box.mx[0] << " " << box.mx[1] << " " << box.mx[2] << " " << box.mx[3] << std::endl;
                    }
                }
            }
        }
    }

    if (!valid) exit(1);

    return valid;
}

}  // namespace mdas
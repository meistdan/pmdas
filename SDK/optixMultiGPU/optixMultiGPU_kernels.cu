#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include "sutil/WorkDistribution.h"

extern "C" __global__ void fillSamples(
        int   gpu_idx,
        int   num_gpus,
        int   width,
        int   height,
        int2* sample_indices )
{
    StaticWorkDistribution wd;
    wd.setRasterSize( width, height );
    wd.setNumGPUs( num_gpus );

    const int sample_idx = blockIdx.x;
    sample_indices[sample_idx] = wd.getSamplePixel( gpu_idx, sample_idx );
}


extern "C" __host__ void fillSamplesCUDA(
        int          num_samples,
        hipStream_t stream,
        int          gpu_idx,
        int          num_gpus,
        int          width,
        int          height,
        int2*        sample_indices )
{
    fillSamples<<<num_samples, 1, 0, stream>>>(
        gpu_idx,
        num_gpus,
        width,
        height,
        sample_indices );
}
